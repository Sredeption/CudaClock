#include "hip/hip_runtime.h"
#include "Reduction.h"

#include <>
#include <hip/hip_runtime.h>


__global__ void timedReductionKernel(const float *input, float *output, clock_t *timer) {
    // __shared__ float shared[2 * blockDim.x];
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    if (tid == 0) timer[bid] = clock();

    // Copy input.
    shared[tid] = input[tid];
    shared[tid + blockDim.x] = input[tid + blockDim.x];

    // Perform reduction to find minimum.
    for (int d = blockDim.x; d > 0; d /= 2) {
        __syncthreads();

        if (tid < d) {
            float f0 = shared[tid];
            float f1 = shared[tid + d];

            if (f1 < f0) {
                shared[tid] = f1;
            }
        }
    }

    // Write result.
    if (tid == 0) output[bid] = shared[0];

    __syncthreads();

    if (tid == 0) timer[bid + gridDim.x] = clock();
}

void timedReduction(const float *input, float *output, clock_t *timer) {
    timedReductionKernel << < NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 * NUM_THREADS >> > (input, output, timer);
}
